#include "hip/hip_runtime.h"
#include <opencv2/core/utility.hpp>
#include "opencv2/video/tracking.hpp"
#include "opencv2/videoio/videoio.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include <signal.h>
#include <syslog.h>
#include <errno.h>

#include <iostream>
#include <ctype.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization
#include <hip/hip_runtime.h>

using namespace cv;
using namespace std;

#define FRAME_WIDTH		    1280
#define FRAME_HEIGHT	    720
#define FRAME_CHANNELS	    3
#define FRAME_SIZE          (FRAME_WIDTH*FRAME_HEIGHT*FRAME_CHANNELS)
#define MAX_GMM_COMPONENTS	3

//GMM parameter
#define ALPHA 0.00005
#define DEF_COVARIANCE  8.0
#define MAX_COVARIANCE  11.0
#define COVARIANCE_THRESHOLD (2.5*2.5)
#define DEF_WEIGHT 0.00005

//Data Structure for GMM
#define THREADS 256 

typedef struct __align__(32)
{
	float3 pixel_mean[FRAME_WIDTH*FRAME_HEIGHT];
	float covariance[FRAME_WIDTH*FRAME_HEIGHT];
	float weight[FRAME_WIDTH*FRAME_HEIGHT];
} gaussian_model;

__device__ 
float3 operator+(const float3 &a, const float3 &b) {

	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ 
float3 operator-(const float3 &a, const float3 &b) {

	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ 
float3 operator*(const float3 &a, const float3 &b) {

	return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}

__device__ 
float3 operator*(const float a, const float3 &b) {

	return make_float3(a*b.x, a*b.y, a*b.z);
}

/*
 * frame is from camera, this function is to initialize the gaussian models
 * It only be invoked once at first
 */
__global__ void
initializeGmm(uchar3* frame, gaussian_model* components)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	components[0].pixel_mean[index] = make_float3(frame[index].x, frame[index].y, frame[index].z);
	components[0].covariance[index] = DEF_COVARIANCE;
	components[0].weight[index] = 1.0;

#pragma unroll
	for (int i = 1; i < MAX_GMM_COMPONENTS; i++) {
		components[i].weight[index] = 0.0;
	}
}

/*
 * frame is from camera, gmm_frame is output filtered by gmm, components always stay on gpu's global memory
 * each invoking performGmm will update the components, return a new gmm_frame
 */
__global__ void
performGmm(const __restrict__ uchar3* frame, unsigned char* gmm_frame, gaussian_model* components)
{
	const int index = blockDim.x * blockIdx.x + threadIdx.x;
	//GMM processing parameter
	bool isMatch = false;
	float sum_of_weight = 0.0;
	float sum_of_square_diff = 0.0;
	float covariance_runtime = 0.0;
	float3 pixel_value;
	float3 pixel_mean;
	float3 pixel_diff;	

	//reset node runtime point
	gaussian_model* current_component = NULL;

	//get BGR value from each pixel
    uchar3 pixel = frame[index];
	pixel_value = make_float3(pixel.x, pixel.y, pixel.z);
	
	//Macthing current pixel for GMM
#pragma unroll
	for(int k = 0; k < MAX_GMM_COMPONENTS; k++) {
		
		current_component = &components[k]; // component from 0 to max
		
		if (current_component->weight[index] == 0) {
			continue;
		}

		if(!isMatch) {
			//Handle matching for each Gaussian Component
			pixel_mean = current_component->pixel_mean[index];
			
			//get diff
			pixel_diff = pixel_value - pixel_mean;
			
			//get covariance for current gaussian model
			covariance_runtime = current_component->covariance[index];

			//get sum of square diff for BGR
			float3 tmp = pixel_diff*pixel_diff;
			sum_of_square_diff = tmp.x + tmp.y + tmp.z;
			
			//judge match or unmatch for current gaussian component
			if( sum_of_square_diff <= (COVARIANCE_THRESHOLD*covariance_runtime*covariance_runtime) )
			{
				//Match current Gaussian component
				//Update weight
				current_component->weight[index] = (1-ALPHA)*(current_component->weight[index]) + ALPHA;
				//Update Gaussian Component
				//Update mean
				pixel_mean = pixel_mean + ALPHA*pixel_diff;
				current_component->pixel_mean[index] = pixel_mean;

				//get new diff
				pixel_diff = pixel_value - pixel_mean;
				
				//update new sum of square_diff	
				float3 tmp = pixel_diff*pixel_diff;
				sum_of_square_diff = tmp.x + tmp.y + tmp.z;

				//Update covariance let Rho = ALPHA
				if (covariance_runtime < MAX_COVARIANCE) {
					covariance_runtime = covariance_runtime + ALPHA*(sum_of_square_diff - covariance_runtime);
					current_component->covariance[index] = covariance_runtime;
				}

				//Set match flag
				isMatch = true;

			}
		}

		if (!isMatch) {
			//UnMatch current Gaussian component
			current_component->weight[index] = (1-ALPHA)*(current_component->weight[index]);
		}

		//get sum of weight
		sum_of_weight += current_component->weight[index];

	} // the end of the k components

	//if there is no match in GMM, delete the least weight gaussian component
	if(!isMatch) {
		
		current_component = &components[0]; 
		int min_component = 0;
		float min_weight = current_component->weight[index];

#pragma unroll
		for (int gg = 1; gg < MAX_GMM_COMPONENTS; gg++) {
			current_component = &components[gg];
			if (current_component->weight[index] < min_weight) {
				min_weight = current_component->weight[index];
				min_component = gg;
			}
		}					
		
		current_component = &components[min_component];
		sum_of_weight -= current_component->weight[index];
		current_component->pixel_mean[index] = pixel_value;
		current_component->covariance[index] = DEF_COVARIANCE;
		current_component->weight[index] = DEF_WEIGHT;

		//record the new sum of weight
		sum_of_weight += current_component->weight[index];
	}
	//normalize the sum of weight to 1, if sum of weight < 0.9 or sum of weight > 1.2
	//do normalization
	if(sum_of_weight < 0.9 || sum_of_weight > 1.2)
	{
		//Normalize the weight for each Gaussian component
#pragma unroll
		for (int gg = 0; gg < MAX_GMM_COMPONENTS; gg++) {
			current_component = &components[gg];
			current_component->weight[index] /= sum_of_weight;
		}
	}
	//judge foreground or background for current pixel and set the result to the gmm_frame
	if(!isMatch){
		//UnMatch any one of background GMM
		gmm_frame[index] = 255;
	} else {
		gmm_frame[index] = 0;
	}
}

/*
 * for allocating device memory
 */
uchar3* d_frame = NULL;	
unsigned char* d_gmm_frame = NULL;
gaussian_model* d_components;

/*
 * for kernel grid and thread num 
 */
int threadsPerBlock = THREADS;
int blocksPerGrid = (FRAME_WIDTH*FRAME_HEIGHT) / threadsPerBlock;

/*
 * cpu invoke gpu kernel to initialize gmm models 
 */
void gpu_initialize_gmm(const Mat &frame)
{
	hipError_t err = hipSuccess;
	
	err = hipMalloc((void **)&d_gmm_frame, FRAME_WIDTH*FRAME_HEIGHT);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device gmm frame (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_frame, FRAME_SIZE);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device frame (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_frame, frame.ptr(0), FRAME_SIZE, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy frame from host to device while initializing (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	if (hipMalloc((void **)&d_components, sizeof(gaussian_model)*MAX_GMM_COMPONENTS) != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate components (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	initializeGmm<<<blocksPerGrid, threadsPerBlock>>>(d_frame, d_components);
	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch initializeGmm kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

/*
 * cpu invoke gpu kernel to perform CUDA-GMM and get output frame filtered by CUDA-GMM
 */
void gpu_perform_gmm(const Mat &frame, Mat &gmm_frame)
{
	hipError_t err = hipSuccess;
	err = hipMemcpy(d_frame, frame.ptr(0), FRAME_SIZE, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy frame from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	performGmm<<<blocksPerGrid, threadsPerBlock>>>(d_frame, d_gmm_frame, d_components);
	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch performGmm kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(gmm_frame.ptr(0), d_gmm_frame, FRAME_WIDTH*FRAME_HEIGHT, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy gmm frame from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

}

int main(int argc, char** argv)
{
	VideoCapture cap;
	Mat frame;

	frame.create(Size(FRAME_WIDTH, FRAME_HEIGHT), CV_8UC1);

	//if ( frame.isContinuous() ) cout << "yes" << endl;
	//Open RGB Camera
	cap.open(0);
	cap.set(cv::CAP_PROP_FRAME_WIDTH, FRAME_WIDTH);
	cap.set(cv::CAP_PROP_FRAME_HEIGHT, FRAME_HEIGHT);

	if( !cap.isOpened() )
	{
		cout << "Can not open camera !!" << endl;
		return -1;
	}

	//read frame
	cap >> frame;

	if( frame.empty() )
	{
		cout << "Can not read data from the Camera !!" << endl;
		return -1;
	}

	gpu_initialize_gmm(frame);

	cout << "frame.cols: " << frame.cols << endl;
	cout << "frame.rows: " << frame.rows << endl;

	for(;;)
	{
		//Get RGB Image
		cap >> frame;

		if( frame.empty() )
		{
			cout << "Can not read data from the Camera !!" << endl;
			return -1;
		}
		
		//GMM output
		Mat gmm_frame;
		gmm_frame.create(frame.size(), frame.type());
		gmm_frame = Mat::zeros(frame.size(), CV_8UC1);
		
		gpu_perform_gmm(frame, gmm_frame);
		//Show the GMM result image
		imshow("GMM", gmm_frame);

		//User Key Input
		char c = waitKey(10);
		if (c == 27) break; // got ESC
	}

	hipError_t err = hipFree(d_frame);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_gmm_frame);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipDeviceReset();

	return 0;
}

